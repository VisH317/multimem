#include "hip/hip_runtime.h"
#include "multicast_alloc.cuh"
#include <vector>
#include <iostream>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void multiMemKernel(hipDeviceptr_t inputMultimemAddr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024*1024*1024) {
        float result = 0;
        // Multimem load-reduce (add) operation from input
        asm volatile(
            "multimem.ld_reduce.relaxed.sys.add.f32 %0, [%1];" 
            : "=f"(result) 
            : "l"(inputMultimemAddr + idx * sizeof(float))
        );
        
        // Multimem store operation to output
        // asm volatile(
        //     "multimem.st.relaxed.sys.f32 [%0], %1;" 
        //     : 
        //     : "l"(outputMultimemAddr + idx * sizeof(float)), "f"(result)
        // );

        printf("inside kernel result: %f\n", result);
    }
}

int main() {
    MC::initializeCuda();

    // initialize multicast vars
    std::vector<int> gpus = { 0, 1 };
    size_t size = 1024 * 1024 * 16; // 1 GB allocation
    hipMemAccessDesc accessDesc; // no modification to this for now, default only

    MC::MCResource mc = MC::createMulticastObject(size, gpus.size());

    for(int gpu : gpus) {
        MC::bindDeviceMemToMulticast(&mc, gpu, accessDesc);
    }

    // memset the allocated addresses
    float fvalue = 1.0;
    int value = *(int*)&fvalue;
    MC::populateMemory(&mc, value);

    // create the multimem address, now stored in mc->mcBuff
    MC::allocateMultimemAddress(&mc, accessDesc);

    std::cout << "Running multimem kernel" << std::endl;
    multiMemKernel<<<1, 1>>>(mc.mcBuff);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    std::cout << "Multimem kernel complete" << std::endl;
}